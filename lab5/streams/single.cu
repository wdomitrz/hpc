#include "hip/hip_runtime.h"
#include "./common/helpers.h"

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x + blockDim.x;
    if (tid < N) {
        int tid1 = (tid + 1) % 256;
        int tid2 = (tid + 2) % 256;
        float aSum = (a[tid] + a[tid1] + a[tid2]) / 3.0f;
        float bSum = (b[tid] + b[tid1] + b[tid2]) / 3.0f;
        c[tid] = (aSum + bSum) / 2;
    }
}

int main(void) {
    hipEvent_t start, stop;
    float elapsedTime;

    hipStream_t stream;
    hipStreamCreate(&stream);

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
        HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

        kernel<<<N / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

        HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));
    }

    HANDLE_ERROR(hipStreamSynchronize(stream));

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time taken: %3.1f ms\n", elapsedTime);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    
    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    HANDLE_ERROR(hipStreamDestroy(stream));
    
    return 0;
}
