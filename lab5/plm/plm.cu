#include "./common/helpers.h"

#define SIZE (10 * 1024 * 1024)

float cuda_malloc_test(int size, bool up) {
    hipEvent_t start, stop;
    int *a, *dev_a;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    a = (int *)malloc(size * sizeof(*a));

    HANDLE_NULL(a);
    HANDLE_ERROR(hipMalloc((void **)&dev_a, size * sizeof(*dev_a)));

    HANDLE_ERROR(hipEventRecord(start, 0));

    for (int i = 0; i < 100; i++) {
        if (up) {
            HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(*dev_a),
                                    hipMemcpyHostToDevice));
        } else {
            HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(*dev_a),
                                    hipMemcpyDeviceToHost));
        }
    }

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    free(a);
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsedTime;
}

float cuda_cudaHostAlloc_test(int size, bool up) {
    hipEvent_t start, stop;
    int *a, *dev_a;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    HANDLE_ERROR(hipHostAlloc(&a, size * sizeof(*a), hipHostMallocDefault));

    HANDLE_NULL(a);
    HANDLE_ERROR(hipMalloc((void **)&dev_a, size * sizeof(*dev_a)));

    HANDLE_ERROR(hipEventRecord(start, 0));

    for (int i = 0; i < 100; i++) {
        if (up) {
            HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(*dev_a),
                                    hipMemcpyHostToDevice));
        } else {
            HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(*dev_a),
                                    hipMemcpyDeviceToHost));
        }
    }

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    hipFree(a);
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsedTime;
}

int main(void) {
    float elapsedTime;
    float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;

    elapsedTime = cuda_malloc_test(SIZE, true);

    printf("Total time for copy up with malloc: %3.1f ms\n", elapsedTime);
    printf("\tMB/s during copy up with malloc:  %3.1f\n",
           MB / (elapsedTime / 1000));

    elapsedTime = cuda_malloc_test(SIZE, false);

    printf("Total time for copy down with malloc: %3.1f ms\n", elapsedTime);
    printf("\tMB/s during copy down with malloc:  %3.1f\n",
           MB / (elapsedTime / 1000));

    elapsedTime = cuda_cudaHostAlloc_test(SIZE, true);

    printf("Total time for copy up with hipMemcpyAsync: %3.1f ms\n",
           elapsedTime);
    printf("\tMB/s during copy up with hipMemcpyAsync:  %3.1f\n",
           MB / (elapsedTime / 1000));

    elapsedTime = cuda_cudaHostAlloc_test(SIZE, false);

    printf("Total time for copy down with hipMemcpyAsync: %3.1f ms\n",
           elapsedTime);
    printf("\tMB/s during copy down with hipMemcpyAsync:  %3.1f\n",
           MB / (elapsedTime / 1000));

    return 0;
}
