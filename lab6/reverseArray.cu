#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 768
#define ARRAY_SIZE THREADS_PER_BLOCK * 1024

static void HandleError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void reverseArray(int *inArray, int *outArray) {
    int inOffset = blockDim.x * blockIdx.x;
    int outOffset = blockDim.x * (gridDim.x - 1 - blockIdx.x);
    int inIndex = inOffset + threadIdx.x;
    int outIndex = outOffset + (blockDim.x - 1 - threadIdx.x);
    outArray[outIndex] = inArray[inIndex];
}

int main(void) {
    int *hostArray;
    int *devInArray, *devOutArray;

    int numBlocks = ARRAY_SIZE / THREADS_PER_BLOCK;

    size_t memSize = ARRAY_SIZE * sizeof(int);
    hostArray = (int *)malloc(memSize);
    HANDLE_ERROR(hipMalloc((void **)&devInArray, memSize));
    HANDLE_ERROR(hipMalloc((void **)&devOutArray, memSize));

    for (int i = 0; i < ARRAY_SIZE; i++) {
        hostArray[i] = i;
    }

    HANDLE_ERROR(
        hipMemcpy(devInArray, hostArray, memSize, hipMemcpyHostToDevice));

    dim3 dimGrid(numBlocks);
    dim3 dimBlock(THREADS_PER_BLOCK);
    reverseArray<<<dimGrid, dimBlock>>>(devInArray, devOutArray);

    hipDeviceSynchronize();

    HANDLE_ERROR(
        hipMemcpy(hostArray, devOutArray, memSize, hipMemcpyDeviceToHost));

    for (int i = 0; i < ARRAY_SIZE; i++) {
        assert(hostArray[i] == ARRAY_SIZE - 1 - i);
    }

    HANDLE_ERROR(hipFree(devInArray));
    HANDLE_ERROR(hipFree(devOutArray));

    free(hostArray);

    printf("Correct!\n");
    return 0;
}
